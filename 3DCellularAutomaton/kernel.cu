#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "CellularAutomaton.h"
#include "cudahelper.h"

struct d_Rule
{
	int* survival;
	int* birth;
	int numStates;
};

// cubeDimension^3 blocks, each block with 27 threads, threads check the current cell's neighbours
__global__ void calculateNextGeneration(int* current, int* next, d_Rule rule)
{
	__shared__ int neighbours[3 * 3 * 3];

	int neighboursIndex = threadIdx.x * 3 * 3 + threadIdx.y * 3 + threadIdx.z;
	neighbours[neighboursIndex] = 0;

	int neighbourCoordinate_x = blockIdx.x + (threadIdx.x - 1);
	int neighbourCoordinate_y = blockIdx.y + (threadIdx.y - 1);
	int neighbourCoordinate_z = blockIdx.z + (threadIdx.z - 1);

	if (neighbourCoordinate_x < 0) {
		neighbourCoordinate_x = gridDim.x - 1;
	}
	if (neighbourCoordinate_y < 0) {
		neighbourCoordinate_y = gridDim.y - 1;
	}
	if (neighbourCoordinate_z < 0) {
		neighbourCoordinate_z = gridDim.z - 1;
	}

	if (neighbourCoordinate_x >= gridDim.x) {
		neighbourCoordinate_x = 0;
	}
	if (neighbourCoordinate_y >= gridDim.y) {
		neighbourCoordinate_y = 0;
	}
	if (neighbourCoordinate_z >= gridDim.z) {
		neighbourCoordinate_z = 0;
	}

	int neighbourLocation =
		neighbourCoordinate_x * (gridDim.x*gridDim.x) +
		neighbourCoordinate_y * (gridDim.x) +
		neighbourCoordinate_z;

	int cellLocation =
		blockIdx.x * (gridDim.x*gridDim.x) +
		blockIdx.y * (gridDim.x) +
		blockIdx.z;

	if (cellLocation != neighbourLocation) {
		if (current[neighbourLocation] == rule.numStates - 1) {
			neighbours[neighboursIndex] = 1;
		}

		__syncthreads();

		if (cellLocation == neighbourLocation) { // run only on one thread
			int neighbourCount = 0;
			for (int i = 0; i < 3 * 3 * 3; i++)
			{
				neighbourCount += neighbours[i];
			}

			if (current[cellLocation] > 0) { // alive cell
				if (rule.survival[neighbourCount] != 1) {
					next[cellLocation] = current[cellLocation] - 1; // aging
				}
			}
			else { // dead cell
				if (rule.birth[neighbourCount] == 1) {
					next[cellLocation] = rule.numStates - 1; // new birth
				}
			}
		}
	}
}


// (cubeDimension/10)^3 blocks with 10*10*10=1000 threads per block, each thread operates on a single cell
__global__ void calculateNextGeneration_updated(int* current, int* next, d_Rule rule)
{
	int xToID = gridDim.x*gridDim.x*blockDim.x*blockDim.x;
	int yToID = gridDim.x*blockDim.x;
	int zToID = 1;

	int dimMax = (gridDim.x - 1)*blockDim.x + (blockDim.x - 1);
	int xMax = dimMax * xToID;
	int yMax = dimMax * yToID;
	int zMax = dimMax * zToID;

	int xCoord = (blockIdx.x*blockDim.x + threadIdx.x);
	int yCoord = (blockIdx.y*blockDim.x + threadIdx.y);
	int zCoord = (blockIdx.z*blockDim.x + threadIdx.z);

	int x = xCoord * xToID;
	int y = yCoord * yToID;
	int z = zCoord * zToID;

	int _x = (x > 0 ? (xCoord - 1)*xToID : xMax); // x minus 1
	int _y = (y > 0 ? (yCoord - 1)*yToID : yMax); // y minus 1
	int _z = (z > 0 ? (zCoord - 1)*zToID : zMax); // z minus 1

	int x_ = (x < xMax ? (xCoord + 1)*xToID : 0); // x plus 1
	int y_ = (y < yMax ? (yCoord + 1)*yToID : 0); // y plus 1
	int z_ = (z < zMax ? (zCoord + 1)*zToID : 0); // z plus 1

	int cellLocation = x + y + z;

	int neighbourCount = 0;
	if (current[_x + _y + _z] == rule.numStates - 1) neighbourCount++;
	if (current[_x + _y + z]  == rule.numStates - 1) neighbourCount++;
	if (current[_x + _y + z_] == rule.numStates - 1) neighbourCount++;
	if (current[_x + y + _z]  == rule.numStates - 1) neighbourCount++;
	if (current[_x + y + z]   == rule.numStates - 1) neighbourCount++;
	if (current[_x + y + z_]  == rule.numStates - 1) neighbourCount++;
	if (current[_x + y_ + _z] == rule.numStates - 1) neighbourCount++;
	if (current[_x + y_ + z]  == rule.numStates - 1) neighbourCount++;
	if (current[_x + y_ + z_] == rule.numStates - 1) neighbourCount++;

	if (current[x + _y + _z]  == rule.numStates - 1) neighbourCount++;
	if (current[x + _y + z]   == rule.numStates - 1) neighbourCount++;
	if (current[x + _y + z_]  == rule.numStates - 1) neighbourCount++;
	if (current[x + y + _z]   == rule.numStates - 1) neighbourCount++;
	if (current[x + y + z_]   == rule.numStates - 1) neighbourCount++;
	if (current[x + y_ + _z]  == rule.numStates - 1) neighbourCount++;
	if (current[x + y_ + z]   == rule.numStates - 1) neighbourCount++;
	if (current[x + y_ + z_]  == rule.numStates - 1) neighbourCount++;

	if (current[x_ + _y + _z] == rule.numStates - 1) neighbourCount++;
	if (current[x_ + _y + z]  == rule.numStates - 1) neighbourCount++;
	if (current[x_ + _y + z_] == rule.numStates - 1) neighbourCount++;
	if (current[x_ + y + _z]  == rule.numStates - 1) neighbourCount++;
	if (current[x_ + y + z]   == rule.numStates - 1) neighbourCount++;
	if (current[x_ + y + z_]  == rule.numStates - 1) neighbourCount++;
	if (current[x_ + y_ + _z] == rule.numStates - 1) neighbourCount++;
	if (current[x_ + y_ + z]  == rule.numStates - 1) neighbourCount++;
	if (current[x_ + y_ + z_] == rule.numStates - 1) neighbourCount++;


	if (current[cellLocation] > 0) { // alive cell
		if (rule.survival[neighbourCount] != 1) {
			next[cellLocation] = current[cellLocation] - 1; // aging
		}
	}
	else { // dead cell
		if (rule.birth[neighbourCount] == 1) {
			next[cellLocation] = rule.numStates - 1; // new birth
		}
	}
}


void CellularAutomaton::updateCells_CUDA()
{
	previousGenerationCells = cells;

	int* d_current;
	int* d_next;

	cudaErrorCheck(hipMalloc(&d_current, sizeof(int) * this->getCellCount()));
	cudaErrorCheck(hipMalloc(&d_next, sizeof(int) * this->getCellCount()));

	cudaErrorCheck(hipMemcpy(d_current, previousGenerationCells.getDataPtr(), sizeof(int) * this->getCellCount(), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_next, cells.getDataPtr(), sizeof(int) * this->getCellCount(), hipMemcpyHostToDevice));

	d_Rule d_rule;
	cudaErrorCheck(hipMalloc(&(d_rule.birth), sizeof(int) * 27));
	cudaErrorCheck(hipMalloc(&(d_rule.survival), sizeof(int) * 27));

	cudaErrorCheck(hipMemcpy(d_rule.birth, rule.d_birth, sizeof(int) * 27, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(d_rule.survival, rule.d_survival, sizeof(int) * 27, hipMemcpyHostToDevice));

	d_rule.numStates = rule.numStates;

	/*dim3 numBlocks(cubeDimension, cubeDimension, cubeDimension);
	dim3 numThreadsPerBlock(3, 3, 3);
	calculateNextGeneration <<< numBlocks, numThreadsPerBlock >>> (d_current, d_next, d_rule);*/

	int blockDim = cubeDimension / 10;
	dim3 numBlocks(blockDim, blockDim, blockDim);
	dim3 numThreadsPerBlock(10, 10, 10); // 1000 threads per block!!!
	calculateNextGeneration_updated <<< numBlocks, numThreadsPerBlock >>> (d_current, d_next, d_rule);


	// Checking failures on kernel
	auto cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA kernel failed: " << hipGetErrorString(cudaStatus) << std::endl;
	}

	cudaErrorCheck(hipDeviceSynchronize());

	cudaErrorCheck(hipMemcpy(cells.getDataPtr(), d_next, sizeof(int) * this->getCellCount(), hipMemcpyDeviceToHost));

	hipFree(d_current);
	hipFree(d_next);
}